#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
void find_moment ( int ** A, int ** B ,int L );


__global__ void moment( int *array1,int *array2,int N,int L,int b,int nBlocks,int blockSize)
{
    int idx= blockIdx.x*blockDim.x*b + threadIdx.x*b;
    __shared__ int array_block[1500];
    int row = idx /L;
    int shared_idx=threadIdx.x*b;
    int left,right;
    int shared_L=blockSize*b;

    for (int i=0;i<b;i++){
        row = (idx+i) /L;
        array_block[shared_idx+i]=array1[idx+i];//kathe thread to diko tou part antigrafei
        array_block[shared_idx+i+ shared_L]=array1[(idx+i+L)%N];
        array_block[shared_idx+i+ 2*shared_L]=array1[(row)?(idx-L+i):(idx+L*(L-1)+i)];
    }

    if(shared_idx==0){
        left=array1[(idx)?((idx - 1)%L + row * L):(L-1)] ;
    }

    if(shared_idx+b==shared_L){
        row = (idx+b-1) /L;
        right=array1[(idx+b)%L + row * L] ;
    }

    __syncthreads();

    for (int i =0;i<b;i++){
        int sum=0;
        int me = array_block[shared_idx+i];
        int n = array_block[shared_idx+i+2*shared_L];//allazei auto se sxesi me ta alla gt o pinakas einai diaforetikos sti shared
        int s = array_block[shared_idx+i+ shared_L];
        int e = (shared_L-shared_idx-i-1)?(array_block[shared_idx+i + 1]):(right) ;
        int w = (shared_idx+i)?( array_block[shared_idx+i - 1]):(left) ;
        sum = sum + me +n + w + s + e ;

        array2[idx+i]= (sum > 0) - (sum < 0);
    }


}

int main(int argc, char *argv[]){

    int ** I;
    int ** J;
    int L = 2000;
    int N= L*L;
    int b=5 ;
    int k = 40;
    int r=0;
    struct timeval startwtime, endwtime;
    double seq_time;
    srand(time(NULL));
    hipEvent_t start,stop;
    float ms;
    int blockSize=500/b;
    int nBlocks = (N/blockSize)/b + (N%(blockSize*b)== 0?0:1);
    printf("nBlocks %d ",nBlocks);
    size_t sharedBytes=10000;
    int *array_host, *array1_device, *array2_device ;
    I=(int**)malloc(L*sizeof(int*));
    J=(int**)malloc(L*sizeof(int*));

    for (int i=0;i<L;i++){
        I[i]=(int*)malloc(L*sizeof(int));
        J[i]=(int*)malloc(L*sizeof(int));
    }

    for (int i=0;i<L;i++) {
        for (int j=0;j<L;j++) {
            r = rand() % 2;
            if(r==0)r=-1;
            I[i][j]=r;
        }
    }

    size_t size = N*sizeof(int);
    array_host=(int*)malloc(size);

    for(int i=0;i<L;i++){
        for(int j=0;j<L;j++)
            array_host[i*L+j]=I[i][j];
    }

    for (int z=0;z<k;z++){
        if(z%2==0){
            find_moment(I,J,L);
        }
        else{
            find_moment(J,I,L);
        }
    }

    /*printf("Final array serial \n");
    if(k%2==0){
        for(int i=0;i<L;i++){
            for(int j=0;j<L;j++)
                printf("%d ",I[i][j]);
            printf("\n");
        }
    }
    else{
        for(int i=0;i<L;i++){
            for(int j=0;j<L;j++)
                printf("%d ",J[i][j]);
            printf("\n");
        }
    }*/

    // ---------------------------------end of serial ---------------------------------------------------------------------------


    hipMalloc((void**)&array1_device, size);
    hipMalloc((void**)&array2_device, size);
    hipMemcpy(array1_device, array_host, size, hipMemcpyHostToDevice);

    gettimeofday (&startwtime, NULL);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i=0;i<k;i++){
        if(i%2==0){
            moment<<<nBlocks,blockSize>>>(array1_device,array2_device,N,L,b,nBlocks,blockSize);
        }
        else{
            moment<<<nBlocks,blockSize>>>(array2_device,array1_device,N,L,b,nBlocks,blockSize);
        }
    }
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);

    gettimeofday (&endwtime, NULL);
    seq_time = (endwtime.tv_sec -startwtime.tv_sec)*1000000L  +endwtime.tv_usec - startwtime.tv_usec ;
    printf("time=%f\n",seq_time);
    printf("time gpu =%f\n",ms);
    if(k%2==0){
        hipMemcpy(array_host, array1_device, size, hipMemcpyDeviceToHost);
    }
    else{
        hipMemcpy(array_host, array2_device, size, hipMemcpyDeviceToHost);
    }

    int counter=0;
    for(int  i=0;i<L;i++){
        for(int j=0;j<L;j++)
            if(I[i][j]!=array_host[i*L+j]){
                counter++;
            }
    }

    printf("counter= %d ",counter);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(array1_device);
    hipFree(array2_device);

    return 0;

}
void find_moment ( int ** A, int ** B ,int L) {
    int sum=0;
    for (int i=0;i<L;i++) {
        for (int j=0;j<L;j++) {
            sum+=A[i][j];
            sum+=A[(i+1)%L][j];
            sum+=A[i?(i-1):(L-1)][j];
            sum+=A[i][(j+1)%L];
            sum+=A[i][j?(j-1):(L-1)];
            if(sum>0)
                B[i][j]=1;
            else
                B[i][j]=-1;
            sum=0;
        }
    }
    return ;
}
